#include<assert.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include "fiddlelink.h"

void *peer_access(void *addr){
        int *tx1 = (int*)addr;
        int *rx1 = (int*)(addr + sizeof(int));
        int tx = *tx1;
        int rx = *rx1;
        printf("tx is %d, rx is %d\n",tx,rx);
        hipSetDevice(tx);
        hipDeviceEnablePeerAccess(rx,0);
}



void pair_stream(int rx, int tx, void* dst, void* src, double size, int type){

	if(type == 3){
		if(dst == src)
			{
                                printf("stream src and dst addtress should be different.\n");
                                exit(0);
                }

                else{
			
			int split = sizeof(char)*8;
			long int chunk = size/split;
			
			hipStream_t stream[2];
			hipStreamCreate(&stream[0]);
			hipStreamCreate(&stream[1]);
			
                        hipDeviceDisablePeerAccess(tx);
                        hipDeviceDisablePeerAccess(rx);

			hipMemcpyAsync(dst,src,chunk,hipMemcpyDeviceToDevice,stream[0]);

			hipDeviceEnablePeerAccess(tx,0);
			hipDeviceEnablePeerAccess(rx,0);

			hipMemcpyAsync(dst,src,chunk,hipMemcpyDeviceToDevice,stream[1]);

			long int address_src = (long long unsigned)src;
			long int address_dst = (long long unsigned)dst;
			
			for(int i = 0; i<split;i++)
			{
				long int address_src1 = address_src+i*chunk;
				long int address_dst1 = address_dst+i*chunk;
				void* src1 = (void*)address_src1;
				void* dst1 = (void*)address_dst1;

                                long int address_src2 = address_src + (i+1)*chunk;
                                long int address_dst2 = address_dst + (i+1)*chunk;
                                void* src2 = (void*)address_src2;
                                void* dst2 = (void*)address_dst2;

                                long int address_src3 = address_src + (i+2)*chunk;
                                long int address_dst3 = address_dst + (i+2)*chunk;
                                void* src3 = (void*)address_src3;
                                void* dst3 = (void*)address_dst3;

				if(hipMemcpyAsync(dst1,src1,chunk,hipMemcpyDeviceToDevice,stream[1])==hipSuccess && hipMemcpyAsync(dst1,src1,chunk,hipMemcpyDeviceToDevice,stream[0])==hipSuccess )
					{

						hipDeviceEnablePeerAccess(tx,0);
						hipMemcpyAsync(dst2,src2,chunk,hipMemcpyDeviceToDevice,stream[1]);					
						hipDeviceDisablePeerAccess(tx);
						hipMemcpyAsync(dst3,src3,chunk,hipMemcpyDeviceToDevice,stream[0]);
						i = i + 1;
					}
				else if (hipMemcpyAsync(dst1,src1,chunk,hipMemcpyDeviceToDevice,stream[0])==hipSuccess && hipMemcpyAsync(dst1,src1,chunk,hipMemcpyDeviceToDevice,stream[1])!=hipSuccess)
					{	
						hipDeviceDisablePeerAccess(tx);
						hipMemcpyAsync(dst2,src2,chunk,hipMemcpyDeviceToDevice,stream[0]);
					}

				else if (hipMemcpyAsync(dst1,src1,chunk,hipMemcpyDeviceToDevice,stream[0])!=hipSuccess && hipMemcpyAsync(dst1,src1,chunk,hipMemcpyDeviceToDevice,stream[1])==hipSuccess)
					{
						hipDeviceEnablePeerAccess(tx,0);
                                                hipMemcpyAsync(dst2,src2,chunk,hipMemcpyDeviceToDevice,stream[1]);
					}
			}

			hipStreamDestroy(stream[0]);
			hipStreamDestroy(stream[1]);
		}


	}


	else if(type == 4){

			hipStream_t stream[2];
                        hipStreamCreate (&stream[0]);
                        hipStreamCreate (&stream[1]);

			
			
                        long int stream0_size = size*8/16;
                        long int stream1_size = size*6/16;
			long int stream2_size = size*2/16;

                        long int address_src = (long long unsigned)src;
                        long int address_src1 = address_src+stream0_size;

                        long int address_dst = (long long unsigned)dst;
                        long int address_dst1 = address_dst+stream0_size;

                        void* src1 = (void*)address_src1;
                        void* dst1 = (void*)address_dst1;

			long int address_src2 = address_src1+stream1_size;
			long int address_dst2 = address_dst1+stream1_size;

			void* src2 = (void*)address_src2;
			void* dst2 = (void*)address_dst2;

			hipMemcpyAsync(dst,src,stream0_size,hipMemcpyDeviceToDevice,stream[0]);
			hipDeviceEnablePeerAccess(tx,0);

                        hipMemcpyAsync(dst1,src1,stream1_size,hipMemcpyDeviceToDevice,stream[1]);
		
                        if (hipMemcpyAsync(dst1,src1,stream1_size,hipMemcpyDeviceToDevice,stream[1])==hipSuccess)

				{
					hipDeviceEnablePeerAccess(tx,0);
					hipMemcpyAsync(dst2,src2,stream2_size,hipMemcpyDeviceToDevice,stream[1]);

				}

/*
			else if(hipMemcpyAsync(dst1,src1,stream1_size,hipMemcpyDeviceToDevice,stream[1])==hipSuccess && hipMemcpyAsync(dst,src,stream0_size,hipMemcpyDeviceToDevice,stream[0])==hipSuccess)
				{	
					long int address_src3 = address_src2+stream2_size*2/3;
                        		long int address_dst3 = address_dst2+stream2_size*2/3;

                        		void* src3 = (void*)address_src3;
                        		void* dst3 = (void*)address_dst3;
					hipMemcpyAsync(dst2,src2,stream2_size,hipMemcpyDeviceToDevice,stream[1]);
					hipDeviceDisablePeerAccess(tx);
					hipMemcpyAsync(dst3,src3,stream2_size/3,hipMemcpyDeviceToDevice,stream[0]);
				}*/		
			else if (hipMemcpyAsync(dst,src,stream0_size,hipMemcpyDeviceToDevice,stream[0])==hipSuccess && hipMemcpyAsync(dst1,src1,stream1_size,hipMemcpyDeviceToDevice,stream[1])!=hipSuccess)
				{

					hipDeviceDisablePeerAccess(tx);
					hipMemcpyAsync(dst2,src2,stream2_size,hipMemcpyDeviceToDevice,stream[0]);
				
				}


		}


        else if(type == 2){
		if(dst == src)
			{
				printf("stream src and dst addtress should be different.\n");
				exit(0);
			}
		else{
			long int stream0_size = size*11/16;
			long int stream1_size = size*5/16;

			long int address_src = (long long unsigned)src;
			long int address_src1 = address_src+stream0_size;

		        long int address_dst = (long long unsigned)dst;
			long int address_dst1 = address_dst+stream0_size;

			void* src1 = (void*)address_src1;
			void* dst1 = (void*)address_dst1;

			hipSetDevice(tx);
                        hipStream_t stream[2];
                        hipStreamCreate (&stream[0]);
                        hipStreamCreate (&stream[1]);
//			hipDeviceDisablePeerAccess(rx);
  //                      hipMemcpyAsync(dst1,src1,stream1_size,hipMemcpyDeviceToDevice,stream[1]);			
				
			hipDeviceEnablePeerAccess(rx,0);
			hipMemcpyAsync(dst,src,stream0_size,hipMemcpyDeviceToDevice,stream[0]);
                        hipDeviceDisablePeerAccess(rx);
                        hipMemcpyAsync(dst1,src1,stream1_size,hipMemcpyDeviceToDevice,stream[1]);
			hipStreamDestroy(stream[0]);
			hipStreamDestroy(stream[1]);	

		}


	}

        else if(type == 1){

		if(dst == src)
			{
				printf("stream src and dst address should be different.\n");
				exit(0);
			}
		else{
                	hipSetDevice(tx);
			hipDeviceEnablePeerAccess(rx,0);
                	hipStream_t stream[1];
                	hipStreamCreate (&stream[0]);

                	hipMemcpyAsync(dst,src,size,hipMemcpyDeviceToDevice,stream[0]);
			hipStreamDestroy(stream[0]);
		}

        }

        else if(type == 0){
	
		if(dst == src)
			{
				printf("stream src and dst address should be different.\n");	
				exit(0);
			}
		else{

			hipSetDevice(tx);
		//	hipDeviceDisablePeerAccess(rx);
                	hipStream_t stream[1];
                	hipStreamCreate(&stream[0]);
                	hipMemcpyAsync(dst,src,size,hipMemcpyDeviceToDevice,stream[0]);
			hipStreamDestroy(stream[0]);
		}

        }

}

